#include "hip/hip_runtime.h"
#include "functional/im2col.h"

/* @brief im2col result shape
 */
size_t im2colResultSize(int n, int c, int h, int w,
                        int kernel_size, int stride, int padding)
{
    int output_height = (h + 2 * padding - kernel_size) / stride + 1;
    int output_width = (w + 2 * padding - kernel_size) / stride + 1;
    int output_size = output_height * output_width;
    return n * c * kernel_size * kernel_size * output_size;
}

Tensor makeIm2colResult(int n, int c, int h, int w,
                        int kernel_size, int stride, int padding)
{
    int im2col_size = im2colResultSize(n, c, w, h, kernel_size, stride, padding);
    checkCppErrorsMsg(im2col_size % 2 != 0, "im2col size should be even");
    auto output = Tensor({im2col_size / 2}, DeviceType::CUDA);
    return output;
}

__global__ static void im2colKernel(float *input, f16 *output,
                                    int n, int c, int h, int w,
                                    int kernel_size, int stride, int padding)
{
    int output_height = (h + 2 * padding - kernel_size) / stride + 1;
    int output_width = (w + 2 * padding - kernel_size) / stride + 1;
    int output_size = output_height * output_width;

    int filter_size = kernel_size * kernel_size;
    int input_channel_size = h * w;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < output_size * filter_size * c * n; i += blockDim.x * gridDim.x)
    {
        // FIXME: unify the order with im2col_naive
        // order: output_h, output_w, filter_h, filter_w, c
        int cur_index = i;
        int cur_n = cur_index % n;
        cur_index /= n;
        int cur_c = cur_index % c;
        cur_index /= c;
        int filter_w = cur_index % kernel_size;
        cur_index /= kernel_size;
        int filter_h = cur_index % kernel_size;
        cur_index /= kernel_size;
        int output_w = cur_index % output_width;
        cur_index /= output_width;
        int output_h = cur_index;

        int index_h = output_h * stride + filter_h - padding;
        int index_w = output_w * stride + filter_w - padding;
        int input_index = (cur_n * c + cur_c) * input_channel_size + index_h * w + index_w;
        // clang-format off
        if (input_index > n * c * h * w) continue;
        // clang-format on

        int output_index = (cur_n * c + cur_c) * filter_size + filter_h * kernel_size + filter_w;
        int output_offset = output_h * output_width + output_w;
        output[output_index * output_size + output_offset] = index_h >= 0 && index_h < h &&
                                                                     index_w >= 0 && index_w < w &&
                                                                     cur_c < c && cur_n < n
                                                                 ? __float2half(input[input_index])
                                                                 : f16(0);
    }
}

/**
 * @brief im2col, performs as the matlab function and at::Tensor function
 *
 * @param input float of shape (N, C, H, W)
 * @param output Output should be shaped as:
 * (N, C * filter_height * filter_width, output_height * output_width)
 *
 * where output_height = (H + 2 * padding - filter_height) / stride + 1
 * and output_width = (W + 2 * padding - filter_width) / stride + 1
 *
 * Data are arranged per channel of columns, this results in factor C.
 */
void im2col(float *input, f16 *output,
            int n, int c, int h, int w,
            int kernel_size, int stride, int padding)
{
    auto single_result_size = im2colResultSize(1, c, h, w, kernel_size, stride, padding);
    // Launch CUDA kernel
    constexpr unsigned long MINIBATCH_SIZE = 2;
    constexpr int N_STREAMS = 8;
    unsigned long minibatches = (n + MINIBATCH_SIZE - 1) / MINIBATCH_SIZE;

    hipStream_t stream[N_STREAMS];
    for (int i = 0; i < N_STREAMS; i++)
        hipStreamCreate(&stream[i]);

    for (unsigned long i = 0; i < minibatches; i++)
    {
        int cur_minibatch_size = std::min(MINIBATCH_SIZE, n - i * MINIBATCH_SIZE);
        int cur_result_size = cur_minibatch_size * single_result_size;

        static const int N_THREADS = 128;
        static const int PER_THREAD = 4;
        dim3 grid_dim((cur_result_size - 1) / N_THREADS / PER_THREAD + 1);
        im2colKernel<<<grid_dim, N_THREADS, 0, stream[i % N_STREAMS]>>>(
            input + i * MINIBATCH_SIZE * c * h * w,
            output + i * MINIBATCH_SIZE * single_result_size,
            cur_minibatch_size, c, h, w,
            kernel_size, stride, padding);
    }

    for (int i = 0; i < N_STREAMS; i++)
    {
        hipStreamSynchronize(stream[i]);
        hipStreamDestroy(stream[i]);
    }
}
