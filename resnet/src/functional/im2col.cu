#include "hip/hip_runtime.h"
/** @file im2col.cu
*/

#include <memory>
#include "common.h"
#include "functional/conv2d.hpp"
#include "functional/macros.h"
#include "mem_pool.h"

using namespace Impl;

/* @brief im2col result shape
 */
size_t im2col_result_size(int N, int C, int H, int W, int filter_height, int filter_width, int stride, int padding) {
  int output_height = (H + 2 * padding - filter_height) / stride + 1;
  int output_width = (W + 2 * padding - filter_width) / stride + 1;
  int output_size = output_height * output_width;
  return N * C * filter_height * filter_width * output_size;
}

std::unique_ptr<float_16[]> create_im2col_result_store_host(int N,
                                                            int C,
                                                            int H,
                                                            int W,
                                                            int filter_height,
                                                            int filter_width,
                                                            int stride,
                                                            int padding) {
  // Allocate memory for im2col result.
  auto im2col_size = im2col_result_size(N, C, H, W, filter_height, filter_width, stride, padding);
  return std::make_unique<float_16[]>(im2col_size);
}

std::unique_ptr<float_16[], decltype(&cudaPooledFree)> create_im2col_result_store_device(int N,
                                                                                   int C,
                                                                                   int H,
                                                                                   int W,
                                                                                   int filter_height,
                                                                                   int filter_width,
                                                                                   int stride,
                                                                                   int padding) {
  auto im2col_size = im2col_result_size(N, C, H, W, filter_height, filter_width, stride, padding);
  float_16 *ptr;
  Impl::cudaPooledMalloc(&ptr, im2col_size * sizeof(float_16));
  return std::unique_ptr<float_16[], decltype(&cudaPooledFree)>(ptr, &cudaPooledFree);
}

__global__ static void im2col_cuda_kernel(const float *input,
                                          float_16 *output,
                                          int N,
                                          int C,
                                          int H,
                                          int W,
                                          int kernel_size,
                                          int stride,
                                          int padding) {
  int output_height = (H + 2 * padding - kernel_size) / stride + 1;
  int output_width = (W + 2 * padding - kernel_size) / stride + 1;
  int output_size = output_height * output_width;

  int filter_size = kernel_size * kernel_size;
  int input_channel_size = H * W;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < output_size * filter_size * C * N; i += blockDim.x * gridDim.x)
  {
    // FIXME: unify the order with im2col_naive
    // order: output_h, output_w, filter_h, filter_w, C
    int cur_index = i;
    int cur_n = cur_index % N;
    cur_index /= N;
    int cur_c = cur_index % C;
    cur_index /= C;
    int filter_w = cur_index % kernel_size;
    cur_index /= kernel_size;
    int filter_h = cur_index % kernel_size;
    cur_index /= kernel_size;
    int output_w = cur_index % output_width;
    cur_index /= output_width;
    int output_h = cur_index;

    int index_h = output_h * stride + filter_h - padding;
    int index_w = output_w * stride + filter_w - padding;
    int input_index = (cur_n * C + cur_c) * input_channel_size + index_h * W + index_w;
    // clang-format off
    if (input_index > N * C * H * W) continue;
    // clang-format on

    int output_index = (cur_n * C + cur_c) * filter_size + filter_h * kernel_size + filter_w;
    int output_offset = output_h * output_width + output_w;
    output[output_index * output_size + output_offset] = index_h >= 0 && index_h < H &&
        index_w >= 0 && index_w < W &&
        cur_c < C && cur_n < N
                                                         ? __float2half(input[input_index])
                                                         : float_16(0);
  }
}

/**
 * @copydoc im2col
 */
static void im2col_device_memory(const float *input,
                                 float_16 *output,
                                 int N,
                                 int C,
                                 int H,
                                 int W,
                                 int filter_height,
                                 int filter_width,
                                 int stride,
                                 int padding) {
  auto single_result_size = im2col_result_size(1, C, H, W, filter_height, filter_width, stride, padding);
  // Launch CUDA kernel
  constexpr unsigned long minibatch_size = 2;
  constexpr int stream_num = 8;
  unsigned long minibatches = (N + minibatch_size - 1) / minibatch_size;

  hipStream_t stream[stream_num];
  for (auto & i : stream) {
    checkCudaErrors(hipStreamCreate(&i));
  }

  for (unsigned long i = 0; i < minibatches; i++) {
    unsigned long curr_minibatch_size = std::min(minibatch_size, (unsigned long) N - i * minibatch_size);
    unsigned long curr_result_size = curr_minibatch_size * single_result_size;

    im2col_cuda_kernel<<<KERNEL_LOOP_BLOCKS(curr_result_size), KERNEL_LOOP_THREADS, 0, stream[i % stream_num]>>>(
        input + i * minibatch_size * C * H * W,
        output + i * minibatch_size * single_result_size,
        (int) curr_minibatch_size,
        C,
        H,
        W,
        filter_height,
        stride,
        padding);
    checkCudaErrors(hipPeekAtLastError());
  }

  for (auto & i : stream) {
    hipStreamSynchronize(i);
    hipStreamDestroy(i);
  }
}

/**
 * @copydoc im2col
 */
static void im2col_host_memory(const float *input,
                               float_16 *output,
                               int N,
                               int C,
                               int H,
                               int W,
                               int filter_height,
                               int filter_width,
                               int stride,
                               int padding) {
  // Copy input to device
  float *input_device;
  hipMalloc(&input_device, sizeof(float) * N * C * H * W);
  hipMemcpy(input_device, input, sizeof(float) * N * C * H * W, hipMemcpyHostToDevice);

  auto result_size = im2col_result_size(N, C, H, W, filter_height, filter_width, stride, padding);
  float_16 *output_device;
  hipMalloc(&output_device, sizeof(float_16) * result_size);

  im2col_device_memory(input_device, output_device, N, C, H, W, filter_height, filter_width, stride, padding);

  // Copy result back to host
  hipMemcpy(output, output_device, sizeof(float_16) * result_size, hipMemcpyDeviceToHost);
  // Free
  hipFree(input_device);
  hipFree(output_device);
}

/**
 * @brief im2col, performs as the matlab function and at::Tensor function
 *
 * @param input float of shape (N, C, H, W)
 * @param output Output should be shaped as:
 * (N, C * filter_height * filter_width, output_height * output_width)
 *
 * where output_height = (H + 2 * padding - filter_height) / stride + 1
 * and output_width = (W + 2 * padding - filter_width) / stride + 1
 *
 * Data are arranged per channel of columns, this results in factor C.
 */
void im2col(const float *input,
            float_16 *output,
            int N,
            int C,
            int H,
            int W,
            int filter_height,
            int filter_width,
            int stride,
            int padding,
            Impl::DeviceType device_type) {
  switch (device_type) {
  case Impl::DeviceType::CPU: return im2col_host_memory(input,
                                                        output,
                                                        N,
                                                        C,
                                                        H,
                                                        W,
                                                        filter_height,
                                                        filter_width,
                                                        stride,
                                                        padding);

  case Impl::DeviceType::CUDA: return im2col_device_memory(input,
                                                           output,
                                                           N,
                                                           C,
                                                           H,
                                                           W,
                                                           filter_height,
                                                           filter_width,
                                                           stride,
                                                           padding);
  }

}

