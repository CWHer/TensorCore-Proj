#include "hip/hip_runtime.h"
/** @file conv2d.cu
*/
#include <memory>

#include "functional/gemm.hpp"
#include "functional/conv2d.hpp"
#include "functional/macros.h"
#include "functional/add.hpp"

#include "common.h"

using namespace Impl;

static void check_cuda_error() {
  hipError_t err = hipPeekAtLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(err));
  }
}

/**
 * @copydoc conv2d
 * @brief Convolutional layer result sizes
 */
int conv2d_output_sizes(int N, int C, int H, int W, int out_channels, int kernel_size, int stride, int padding) {
  auto shape = conv2d_result_shape(N, C, H, W, out_channels, kernel_size, stride, padding);
  return shape[0] * shape[1] * shape[2] * shape[3];
}

/**
 * @copydoc conv2d
 * @brief Convolutional layer result shape
 */
std::vector<int> conv2d_result_shape(int N,
                                     int C,
                                     int H,
                                     int W,
                                     int out_channels,
                                     int kernel_size,
                                     int stride,
                                     int padding) {
  int output_height = (H + 2 * padding - kernel_size) / stride + 1;
  int output_width = (W + 2 * padding - kernel_size) / stride + 1;
  return {N, out_channels, output_height, output_width};
}

/** @brief Convolutional layer forward propagation.
 *
 * @param input Input float, row major (organizes at last element), of shape (N, C, H, W).
 * @param output Result float, row major (organizes at last element), of shape (N, out_channels, H_out, W_out).
 *  where H_out = floor((H + 2 * padding - kernel_size) / stride) + 1
 *  and W_out = floor((W + 2 * padding - kernel_size) / stride) + 1
 *  @param weight Weight float_16, row major (organizes at last element), of shape (out_channels, C, kernel_size, kernel_size).
 *  @param bias Bias float, row major (organizes at last element), in shape of (out_channels).
 *  @param N Batch size.
 *  @param C Number of input channels.
 *  @param H Height of input.
 *  @param W Width of input.
 *  @param kernel_size Size of kernel (kernel_size x kernel_size).
 *  @param stride Stride of convolution.
 *  @param padding Padding of convolution.
 */
void conv2d(const float *input,
            float *output,
            const float_16 *weight,
            const float *bias,
            int N,
            int C,
            int H,
            int W,
            int out_channels,
            int kernel_size,
            int stride,
            int padding,
            Impl::DeviceType device_type) {
  int output_height = (H + 2 * padding - kernel_size) / stride + 1;
  int output_width = (W + 2 * padding - kernel_size) / stride + 1;
  int conv_result_size = output_height * output_width;
  int expanded_kernel_width = C * kernel_size * kernel_size;

  if (bias) {
    throw std::runtime_error("Conv2d with bias not implemented");
  }

  int batched_n = 128;

  if (device_type == Impl::DeviceType::CUDA) {
    auto total_batch_size = (N + batched_n - 1) / batched_n;
    if (total_batch_size == 1) {
      auto im2col_result = create_im2col_result_store_device(N, C, H, W, kernel_size, kernel_size, stride, padding);
      im2col(input, im2col_result.get(), N, C, H, W, kernel_size, kernel_size, stride, padding, device_type);
      gemm_batched_B(weight,
                     im2col_result.get(),
                     output,
                     out_channels,
                     conv_result_size,
                     expanded_kernel_width,
                     N,
                     GEMM::Major::row_major,
                     device_type);
    } else {
      auto im2col_result =
          create_im2col_result_store_device(batched_n, C, H, W, kernel_size, kernel_size, stride, padding);
      for (int i = 0; i < total_batch_size; ++i) {
        auto batch_size = std::min(batched_n, N - i * batched_n);
        if (batch_size == 0) {
          break;
        }
        if (batch_size != batched_n) {
          im2col_result =
              create_im2col_result_store_device(batch_size, C, H, W, kernel_size, kernel_size, stride, padding);
        }
        im2col(input + i * batched_n * C * H * W,
               im2col_result.get(),
               batch_size,
               C,
               H,
               W,
               kernel_size,
               kernel_size,
               stride,
               padding,
               device_type);
        gemm_batched_B(weight,
                       im2col_result.get(),
                       output + i * batched_n * out_channels * conv_result_size,
                       out_channels,
                       conv_result_size,
                       expanded_kernel_width,
                       batch_size,
                       GEMM::Major::row_major,
                       device_type);
      }
    }

    //Impl::cudaPooledMalloc(&bias_expanded, out_channels * conv_result_size * sizeof(float));
  } else {
    auto im2col_result = create_im2col_result_store_host(N, C, H, W, kernel_size, kernel_size, stride, padding);
    // After im2col, im2col_result is of shape (N, C * kernel_size * kernel_size, H_out * W_out)
    im2col(input, im2col_result.get(), N, C, H, W, kernel_size, kernel_size, stride, padding, device_type);

    /**
     * FIXME: Current im2col implementation, according to pytorch, moves N to the top level, which is not efficient.
     * the N should be moved to make sure the number of column be C * kernel_size * kernel_size, then we can utilize
     * a large GEMM and then swap the dimensions to the right shape.
     */
    gemm_batched_B(weight,
                   im2col_result.get(),
                   output,
                   out_channels,
                   conv_result_size,
                   expanded_kernel_width,
                   N,
                   GEMM::Major::row_major,
                   device_type);
  }

  check_cuda_error();
}

