#include "hip/hip_runtime.h"
#include <cfloat>
#include "common.h"

template <int block_size>
__global__ void deviceMaxPool2dKernel(float *input_data, int height, int width,
                                      float *output_data, int out_height, int out_width,
                                      int kernel_size, int padding, int stride)
{
    // HACK: NOTE: Max-pooling uses implicit negative infinity padding,
    //  not zero-padding as indicated in documentation
    // https://github.com/pytorch/pytorch/issues/33384
    int input_grid_offset = blockIdx.x * height * width;
    int output_grid_offset = blockIdx.x * out_height * out_width;

    int row = block_size * threadIdx.y;
    int col = block_size * threadIdx.x;
    for (int i = 0; i < block_size; i++)
        for (int j = 0; j < block_size; j++)
            if (row + i < out_height && col + j < out_width)
            {
                float value = -FLT_MAX;
                for (int x = 0; x < kernel_size; ++x)
                    for (int y = 0; y < kernel_size; ++y)
                    {
                        int input_row = (row + i) * stride + x - padding;
                        int input_col = (col + j) * stride + y - padding;
                        if (0 <= input_row && input_row < height &&
                            0 <= input_col && input_col < width)
                        {
                            float input_value = input_data[input_grid_offset +
                                                           input_row * width + input_col];
                            value = max(value, input_value);
                        }
                    }
                output_data[output_grid_offset +
                            (row + i) * out_width + (col + j)] = value;
            }
}

// TODO: tiling and leverage cache
template <int block_size>
__global__ void deviceAvgPool2dKernel(const float *input_data, int height, int width,
                                      float *output_data, int out_height, int out_width,
                                      int kernel_size, int padding, int stride)
{
    // NOTE: B/2 (block) x 128 (thread) x H x W (within)
    int input_grid_offset = blockDim.x * height * width;
    int output_grid_offset = blockDim.x * out_height * out_width;
    int input_thread_offset = height * width;
    int output_thread_offset = out_height * out_width;

    input_data += blockIdx.x * input_grid_offset +
                  threadIdx.x * input_thread_offset;
    output_data += blockIdx.x * output_grid_offset +
                   threadIdx.x * output_thread_offset;

    int output_offset = 0;
    float r = 1.0f / (kernel_size * kernel_size);
    for (int i = kernel_size - 1 - padding; i < height + padding; i += stride)
        for (int j = kernel_size - 1 - padding; j < width + padding; j += stride)
        {
            float ret = 0;
            for (int x = i - kernel_size + 1; x <= i; x++)
                for (int y = j - kernel_size + 1; y <= j; y++)
                {
                    float value = 0 <= x && x < height &&
                                          0 <= y && y < width
                                      ? input_data[x * width + y]
                                      : 0; // zero padding
                    ret += value;
                }
            output_data[output_offset++] = ret * r;
        }
}

void maxpool2d(const float *input_data,
               int batch_size,
               int num_channels,
               int height,
               int width,
               float *output_data,
               int out_height,
               int out_width,
               int kernel_size,
               int padding,
               int stride)
{
    static const int N_THREADS = 16;
    dim3 grid_dim(batch_size * num_channels);
    dim3 block_dim(N_THREADS, N_THREADS);
#if DEBUG
    checkCppErrorsMsg(height != width || out_height != out_width,
                      "Only square images are supported");
#endif
    deviceMaxPool2dKernel<4><<<grid_dim, block_dim>>>(
        (float *)input_data, height, width, output_data, out_height, out_width,
        kernel_size, padding, stride);
}

void avgpool2d(const float *input_data,
               int batch_size,
               int num_channels,
               int height,
               int width,
               float *output_data,
               int out_height,
               int out_width,
               int kernel_size,
               int padding,
               int stride){
  static const int N_THREADS = 128;
  dim3 block_dim(N_THREADS);
  dim3 grid_dim(batch_size * num_channels / N_THREADS);
  deviceAvgPool2dKernel<32><<<grid_dim, block_dim>>>(
      input_data, height, width, output_data, out_height, out_width,
      kernel_size, padding, stride);
}
