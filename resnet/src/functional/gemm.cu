#include "hip/hip_runtime.h"
/** @file gemm.cu
*/
#include <mma.h>
#include <hipblas.h>
#include "common.h"
#include "functional/macros.h"
#include "functional/gemm.hpp"

using namespace nvcuda;
using namespace Impl;

static void check_cuda_error() {
  hipError_t err = hipPeekAtLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(err));
  }
}

static const int warp_size = 32;

// For Volta architecture, only FP16 of 16x16x16 is supported.
static const int volta_m_factor = 16;
static const int volta_n_factor = 16;
static const int volta_k_factor = 16;

template<int block_col_warps, int block_row_warps> static __global__ void gemm_naive_kernel(const float_16 *A,
                                                                                            const float_16 *B,
                                                                                            float_32 *Result,
                                                                                            size_t M,
                                                                                            size_t N,
                                                                                            size_t K) {

  const int block_threads = warp_size * (block_row_warps * block_col_warps);

  constexpr int tile_m = block_row_warps * volta_m_factor;
  constexpr int tile_n = block_col_warps * volta_n_factor;
  constexpr int tile_k = tile_m;

  __shared__ float_16 As[tile_k][tile_m];
  __shared__ float_16 Bs[tile_n][tile_k];

  auto tid = threadIdx.y * blockDim.x + threadIdx.x;

  auto aRow = blockIdx.x * tile_m;
  auto bCol = blockIdx.y * tile_n;

  wmma::fragment<wmma::matrix_a, volta_m_factor, volta_n_factor, volta_k_factor, float_16, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, volta_m_factor, volta_n_factor, volta_k_factor, float_16, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, volta_m_factor, volta_n_factor, volta_k_factor, float_32> result_frag;
  wmma::fill_fragment(result_frag, 0.0f);

  for (int k = 0; k < K; k += tile_k) {
    // Parallel loading
    for (int i = 0; i < tile_m * tile_k; i += block_threads) {
      auto idx = (tid + i);

      auto As_row = idx % tile_m;
      auto As_col = idx / tile_m;
      auto Bs_row = idx % tile_k;
      auto Bs_col = idx / tile_k;

      if (aRow + As_row < M && k + As_col < K) {
        As[As_col][As_row] = A[(k + As_col) * M + aRow + As_row];
      } else {
        As[As_col][As_row] = 0;
      }

      if (k + Bs_row < K && bCol + Bs_col < N) {
        Bs[Bs_col][Bs_row] = B[(bCol + Bs_col) * K + k + Bs_row];
      } else {
        Bs[Bs_col][Bs_row] = 0;
      }

    }
    __syncthreads();

    for (int i = 0; i < tile_k; i += volta_k_factor) {
      auto As_offset = i * tile_m + volta_m_factor * (threadIdx.x / warp_size);
      auto Bs_offset = volta_n_factor * threadIdx.y * tile_k + i;

      wmma::load_matrix_sync(a_frag, (half *) As + As_offset, tile_m);
      wmma::load_matrix_sync(b_frag, (half *) Bs + Bs_offset, tile_k);

      wmma::mma_sync(result_frag, a_frag, b_frag, result_frag);
    }
  }

  auto cRow = (blockIdx.x * blockDim.x + threadIdx.x) / warp_size * volta_m_factor;
  auto cCol = (blockIdx.y * blockDim.y + threadIdx.y) * volta_n_factor;
  auto c_offset = cRow + cCol * M;

  if (cRow < M && cCol < N) {
    wmma::store_matrix_sync(Result + c_offset, result_frag, M, wmma::mem_col_major);
  }
}

template<int block_col_warps, int block_row_warps> static void gemm_naive_caller(const float_16 *A,
                                                                                 const float_16 *B,
                                                                                 float_32 *Result,
                                                                                 size_t M,
                                                                                 size_t N,
                                                                                 size_t K,
                                                                                 hipStream_t &stream) {
  constexpr int tile_m = block_row_warps * volta_m_factor;
  constexpr int tile_n = block_col_warps * volta_n_factor;

  dim3 grid((M + (tile_m - 1)) / tile_m, (N + (tile_n - 1)) / tile_n);
  dim3 block(block_row_warps * warp_size, block_col_warps);

  gemm_naive_kernel<block_col_warps, block_row_warps><<<grid, block, 0, stream>>>(A, B, Result, M, N, K);
}

template<typename T, hipMemcpyKind memcpy_kind, bool require_copy> static T *gemm_padding_col_major(const T *source,
                                                                                                     size_t row,
                                                                                                     size_t col,
                                                                                                     size_t pad_row,
                                                                                                     size_t pad_col,
                                                                                                     hipStream_t &stream) {
  if ((col == pad_col) && (row == pad_row)
      && (memcpy_kind == hipMemcpyHostToHost || memcpy_kind == hipMemcpyDeviceToDevice))
    return (T *) source;

  T *padded;
  checkCudaErrors(cudaMallocAsyncIfAvailable(&padded, sizeof(T) * pad_col * pad_row, stream));

  if (require_copy) {
    checkCudaErrors(hipMemsetAsync((void *) padded, 0, sizeof(T) * pad_col * pad_row, stream));
    checkCudaErrors(hipMemcpy2DAsync(padded,
                                      sizeof(T) * pad_col,
                                      source,
                                      sizeof(T) * col,
                                      sizeof(T) * col,
                                      row,
                                      memcpy_kind,
                                      stream));
  }

  return padded;
}

template<typename T, hipMemcpyKind memcpy_kind> static void gemm_unpad_col_major(T *source,
                                                                                  T *padded,
                                                                                  size_t row,
                                                                                  size_t col,
                                                                                  size_t pad_row,
                                                                                  size_t pad_col,
                                                                                  hipStream_t &stream) {
  if (source == padded && (memcpy_kind == hipMemcpyHostToHost || memcpy_kind == hipMemcpyDeviceToDevice)) {
    return;
  }

  if ((col == pad_col) && (row == pad_row)) {
    checkCudaErrors(hipMemcpyAsync(source, padded, sizeof(T) * col * row, memcpy_kind, stream));
  } else {
    checkCudaErrors(hipMemcpy2DAsync(source,
                                      sizeof(T) * col,
                                      padded,
                                      sizeof(T) * pad_col,
                                      sizeof(T) * col,
                                      row,
                                      memcpy_kind,
                                      stream));
  }
}

static void gemm_device_memory(const float_16 *A,
                               const float_16 *B,
                               float_32 *Result,
                               size_t M,
                               size_t N,
                               size_t K,
                               hipStream_t &stream) {
  float_16 *padded_A;
  float_16 *padded_B;
  float_32 *padded_C;

  // If M and N are not by 16, we need to pad them.
  auto padded_M = (M + (volta_m_factor - 1)) / volta_m_factor * volta_m_factor;
  auto padded_N = (N + (volta_n_factor - 1)) / volta_n_factor * volta_n_factor;

  // Copy A and B by padding to device
  // B needs padding, with a leading dimension of N

  padded_A = gemm_padding_col_major<float_16, hipMemcpyDeviceToDevice, true>(A, K, M, K, padded_M, stream);
  padded_B = gemm_padding_col_major<float_16, hipMemcpyDeviceToDevice, true>(B, N, K, padded_N, K, stream);
  padded_C =
      gemm_padding_col_major<float_32, hipMemcpyDeviceToDevice, false>(Result, N, M, padded_N, padded_M, stream);

  // Fixme: this template parameter is adjustable
  gemm_naive_caller<4, 4>(padded_A, padded_B, padded_C, padded_M, padded_N, K, stream);

  if (padded_C != Result) {
    gemm_unpad_col_major<float_32, hipMemcpyDeviceToDevice>(Result, padded_C, N, M, padded_N, padded_M, stream);

  }

#if not CUDA_MALLOC_ASYNC
    checkCudaErrors(hipStreamSynchronize(stream));
#endif

  if (padded_A != A)
    checkCudaErrors(cudaFreeAsyncIfAvailable(padded_A, stream));

  if (padded_B != B)
    checkCudaErrors(cudaFreeAsyncIfAvailable(padded_B, stream));

  if (padded_C != Result) {
    checkCudaErrors(cudaFreeAsyncIfAvailable(padded_C, stream));
  }

}

static void gemm_host_memory(const float_16 *A,
                             const float_16 *B,
                             float_32 *Result,
                             size_t M,
                             size_t N,
                             size_t K,
                             hipStream_t &stream) {
  float_16 *padded_A;
  float_16 *padded_B;
  float_32 *padded_C;

  // If M and N are not by 16, we need to pad them.
  auto padded_M = (M + (volta_m_factor - 1)) / volta_m_factor * volta_m_factor;
  auto padded_N = (N + (volta_n_factor - 1)) / volta_n_factor * volta_n_factor;

  // Copy A and B by padding to device
  // Copy A and B by padding to device
  // B needs padding, with a leading dimension of N
  padded_A = gemm_padding_col_major<float_16, hipMemcpyHostToDevice, true>(A, K, M, K, padded_M, stream);
  padded_B = gemm_padding_col_major<float_16, hipMemcpyHostToDevice, true>(B, N, K, padded_N, K, stream);
  padded_C = gemm_padding_col_major<float_32, hipMemcpyHostToDevice, false>(Result, N, M, padded_N, padded_M, stream);

  // Fixme: this template parameter is adjustable
  gemm_device_memory(padded_A, padded_B, padded_C, padded_M, padded_N, K, stream);

  gemm_unpad_col_major<float_32, hipMemcpyDeviceToHost>(Result, padded_C, N, M, padded_N, padded_M, stream);

#if not CUDA_MALLOC_ASYNC
  checkCudaErrors(hipStreamSynchronize(stream));
#endif

  checkCudaErrors(cudaPooledFree(padded_A));
  checkCudaErrors(cudaPooledFree(padded_B));
  checkCudaErrors(cudaPooledFree(padded_C));
}

void gemm_stream(const float_16 *A,
                 const float_16 *B,
                 float_32 *C,
                 size_t M,
                 size_t N,
                 size_t K,
                 const GEMM::Major major,
                 const Impl::DeviceType device_type,
                 hipStream_t &stream) {
  switch (device_type) {
  case Impl::DeviceType::CPU:
    switch (major) {
    case GEMM::Major::col_major:gemm_host_memory(A, B, C, M, N, K, stream);
      break;
    case GEMM::Major::row_major:gemm_host_memory(B, A, C, N, M, K, stream);
      break;
    }
    break;
  case Impl::DeviceType::CUDA:
    switch (major) {
    case GEMM::Major::col_major:gemm_device_memory(A, B, C, M, N, K, stream);
      break;
    case GEMM::Major::row_major:gemm_device_memory(B, A, C, N, M, K, stream);
      break;
    }
    break;
  }
}


