#include "hip/hip_runtime.h"
/** @file gemm.cpp
*/
#include <mma.h>
#include <hipblas.h>
#include "common.h"

using namespace nvcuda;

static void check_cuda_error() {
  hipError_t err = hipPeekAtLastError();
  if (err != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(err));
  }
}

static const int warp_size = 32;

// For Volta architecture, only FP16 of 16x16x16 is supported.
static const int volta_m_factor = 16;
static const int volta_n_factor = 16;
static const int volta_k_factor = 16;

template<int block_col_warps, int block_row_warps> static __global__ void gemm_naive_kernel(const float_16 *A,
                                                                                            const float_16 *B,
                                                                                            float_32 *Result,
                                                                                            size_t M,
                                                                                            size_t N,
                                                                                            size_t K) {

  const int block_threads = warp_size * (block_row_warps * block_col_warps);

  constexpr int tile_m = block_row_warps * volta_m_factor;
  constexpr int tile_n = block_col_warps * volta_n_factor;
  constexpr int tile_k = tile_m;

  __shared__ float_16 As[tile_k][tile_m];
  __shared__ float_16 Bs[tile_n][tile_k];

  auto tid = threadIdx.y * blockDim.x + threadIdx.x;

  auto aRow = blockIdx.x * tile_m;
  auto bCol = blockIdx.y * tile_n;

  wmma::fragment<wmma::matrix_a, volta_m_factor, volta_n_factor, volta_k_factor, float_16, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, volta_m_factor, volta_n_factor, volta_k_factor, float_16, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, volta_m_factor, volta_n_factor, volta_k_factor, float_32> result_frag;
  wmma::fill_fragment(result_frag, 0.0f);

  for (int k = 0; k < K; k += tile_k) {
    // Parallel loading
    for (int i = 0; i < tile_m * tile_k; i += block_threads) {
      auto idx = (tid + i);

      auto As_row = idx % tile_m;
      auto As_col = idx / tile_m;
      auto Bs_row = idx % tile_k;
      auto Bs_col = idx / tile_k;

      if (aRow + As_row < M && k + As_col < K) {
        As[As_col][As_row] = A[(k + As_col) * M + aRow + As_row];
      } else {
        As[As_col][As_row] = 0;
      }

      if (k + Bs_row < K && bCol + Bs_col < N) {
        Bs[Bs_col][Bs_row] = B[(bCol + Bs_col) * K + k + Bs_row];
      } else {
        Bs[Bs_col][Bs_row] = 0;
      }

    }
    __syncthreads();

    for (int i = 0; i < tile_k; i += volta_k_factor) {
      auto As_offset = i * tile_m + volta_m_factor * (threadIdx.x / warp_size);
      auto Bs_offset = volta_n_factor * threadIdx.y * tile_k + i;

      wmma::load_matrix_sync(a_frag, (half *) As + As_offset, tile_m);
      wmma::load_matrix_sync(b_frag, (half *) Bs + Bs_offset, tile_k);

      wmma::mma_sync(result_frag, a_frag, b_frag, result_frag);
    }
  }

  auto cRow = (blockIdx.x * blockDim.x + threadIdx.x) / warp_size * volta_m_factor;
  auto cCol = (blockIdx.y * blockDim.y + threadIdx.y) * volta_n_factor;
  auto c_offset = cRow + cCol * M;

  if (cRow < M && cCol < N) {
    wmma::store_matrix_sync(Result + c_offset, result_frag, M, wmma::mem_col_major);
  }
}


template<int block_col_warps, int block_row_warps>
void gemm_naive_caller(const float_16 *A, const float_16 *B, float_32 *Result, size_t M, size_t N, size_t K) {
  constexpr int tile_m = block_row_warps * volta_m_factor;
  constexpr int tile_n = block_col_warps * volta_n_factor;

  dim3 grid((M + (tile_m - 1)) / tile_m, (N + (tile_n - 1)) / tile_n);
  dim3 block(block_row_warps * warp_size, block_col_warps);

  check_cuda_error();

  gemm_naive_kernel<block_col_warps, block_row_warps><<<grid, block>>>(A, B, Result, M, N, K);

  check_cuda_error();
}


void gemm_naive(const float_16 *A, const float_16 *B, float_32 *Result, size_t M, size_t N, size_t K) {
  float_16 *d_A, *d_B;
  float_32 *d_C;

  hipMalloc(&d_A, M * K * sizeof(float_16));
  hipMalloc(&d_B, K * N * sizeof(float_16));
  hipMalloc(&d_C, M * N * sizeof(float_32));

  hipMemcpy(d_A, A, M * K * sizeof(float_16), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, K * N * sizeof(float_16), hipMemcpyHostToDevice);

  // Fixme: this template parameter is adjustable
  gemm_naive_caller<4, 4>(d_A, d_B, d_C, M, N, K);

  hipMemcpy(Result, d_C, M * N * sizeof(float_32), hipMemcpyDeviceToHost);
  check_cuda_error();

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  check_cuda_error();
}
