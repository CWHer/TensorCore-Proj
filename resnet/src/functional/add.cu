#include "hip/hip_runtime.h"
/** @file add.cu
*/

#include "functional/macros.hpp"
#include "functional/add.hpp"
#include "common.hpp"

__global__ static void cuda_add_(float *RESTRICT Result, const float *RESTRICT adder, size_t length) {
  CUDA_KERNEL_LOOP(i, length) {
    if (i < length) {
      Result[i] += adder[i];
    }
  }
}

__global__ static void cuda_add(float *RESTRICT Result,
                                const float *RESTRICT adder_a,
                                const float *RESTRICT adder_b,
                                size_t length) {
  CUDA_KERNEL_LOOP(i, length) {
    if (i < length) {
      Result[i] = adder_a[i] + adder_b[i];
    }
  }
}

__global__ static void cuda_relu_(float *RESTRICT Result, size_t length) {
  CUDA_KERNEL_LOOP(i, length) {
    if (i < length) {
      Result[i] = Result[i] > 0.0f ? Result[i] : 0.0f;
    }
  }
}

__global__ static void cuda_add_relu_(float *RESTRICT Result, const float *RESTRICT adder, size_t length) {
  CUDA_KERNEL_LOOP(i, length) {
    if (i < length) {
      Result[i] += adder[i];
      Result[i] = Result[i] > 0.0f ? Result[i] : 0.0f;
    }
  }
}

void add_(float *RESTRICT Result,
          const float *RESTRICT adder,
          size_t length,
          Impl::DeviceType device_type,
          hipStream_t stream) {
  switch (device_type) {
  case Impl::DeviceType::CPU: {
    for (size_t i = 0; i < length; ++i) {
      Result[i] += adder[i];
    }
  }
    break;
  case Impl::DeviceType::CUDA: cuda_add_<<<KERNEL_LOOP_BLOCKS(length), KERNEL_LOOP_THREADS, 0, stream>>>(Result,
                                                                                                         adder,
                                                                                                         length);
    break;
  }
}

[[maybe_unused]]
void add(float *Result, const float *adder_a, const float *adder_b, int length, Impl::DeviceType device_type) {
  switch (device_type) {
  case Impl::DeviceType::CPU: {
    for (int i = 0; i < length; ++i) {
      Result[i] = adder_a[i] + adder_b[i];
    }
  }
    break;
  case Impl::DeviceType::CUDA: cuda_add<<<KERNEL_LOOP_BLOCKS(length), KERNEL_LOOP_THREADS>>>(Result,
                                                                                             adder_a,
                                                                                             adder_b,
                                                                                             length);
    break;
  }
}

void relu_(float *RESTRICT Result, size_t length, Impl::DeviceType device_type) {
  switch (device_type) {
  case Impl::DeviceType::CPU: {
    for (size_t i = 0; i < length; ++i) {
      Result[i] = Result[i] > 0.0f ? Result[i] : 0.0f;
    }
  }
    break;
  case Impl::DeviceType::CUDA: cuda_relu_<<<KERNEL_LOOP_BLOCKS(length), KERNEL_LOOP_THREADS>>>(Result, length);
    break;
  }
}

void add_relu_(float *RESTRICT Result, const float *RESTRICT adder, size_t length, Impl::DeviceType device_type) {
  switch (device_type) {
  case Impl::DeviceType::CPU: {
    for (size_t i = 0; i < length; ++i) {
      Result[i] += adder[i];
      Result[i] = Result[i] > 0.0f ? Result[i] : 0.0f;
    }
  }
    break;
  case Impl::DeviceType::CUDA: cuda_add_relu_<<<KERNEL_LOOP_BLOCKS(length), KERNEL_LOOP_THREADS>>>(Result,
                                                                                                   adder,
                                                                                                   length);
    break;
  }
}
