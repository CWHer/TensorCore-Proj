#include "hip/hip_runtime.h"
/** @file linear.cu
*/

// Linear layer is performed by trivial GEMM

#include "functional/gemm.hpp"
#include "functional/add.hpp"
#include "functional/linear.hpp"
#include "functional/macros.h"

/**
 * @brief Linear layer implementation.
 *
 * @param input Input tensor of shape (batch_size, input_channel)
 * @param output Output tensor of shape (batch_size, output_channel)
 * @param weight Weight tensor of shape (input_channel, output_channel)
 * @param bias Bias tensor of shape (output_channel)
 * @param input_channel
 * @param output_channel
 */
void linear(const float_16 *input,
            float *output,
            const float_16 *weight,
            const float *bias,
            int batch_size,
            int input_channel,
            int output_channel,
            Impl::DeviceType device_type) {
  switch (device_type) {
  case Impl::DeviceType::CPU : {
    gemm(input,
         weight,
         output,
         batch_size,
         output_channel,
         input_channel,
         GEMM::Major::row_major,
         Impl::DeviceType::CPU);
  }
    break;
  case Impl::DeviceType::CUDA : {
    gemm(input,
         weight,
         output,
         batch_size,
         output_channel,
         input_channel,
         GEMM::Major::row_major,
         Impl::DeviceType::CUDA);
  }
    break;
  }

  constexpr int stream_num = 8;
  hipStream_t streams[stream_num];
  for (int i = 0; i < stream_num; ++i) {
    hipStreamCreate(&streams[i]);
  }

  for (int batch = 0; batch < batch_size; ++batch) {
    add_(&output[batch * output_channel], bias, output_channel, device_type, streams[batch % stream_num]);
  }

  for (int i = 0; i < stream_num; ++i) {
    hipStreamSynchronize(streams[i]);
    hipStreamDestroy(streams[i]);
  }
}

__global__ void transpose_kernel(const float *input, float_16 *output, int m, int n) {
  CUDA_KERNEL_LOOP(index, m * n) {
    int i = index / n;
    int j = index % n;
    if (i < m && j < n) {
      output[j * m + i] = __float2half(input[i * n + j]);
    }
  }
}

void prepare_linear_weight(const float *input, float_16 *output, int row, int col, Impl::DeviceType device_type) {
  switch (device_type) {
  case Impl::DeviceType::CPU : {
    for (int i = 0; i < row; ++i) {
      for (int j = 0; j < col; ++j) {
        output[j * row + i] = __float2half(input[i * col + j]);
      }
    }
  }
    break;
  case Impl::DeviceType::CUDA : {
    transpose_kernel<<<KERNEL_LOOP_BLOCKS(row * col), KERNEL_LOOP_THREADS>>>(input, output, row, col);
  }
    break;
  }
}
