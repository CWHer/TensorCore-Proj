#include "hip/hip_runtime.h"
/**
 * @file test/gemm.cu
 * @brief Test GEMM functionality.
 */

#include <gtest/gtest.h>
#include "functional/gemm.hpp"
#include <random>
#include <hipblas.h>

#ifndef RANDOM_SEED
#define RANDOM_SEED std::random_device{}()
#endif

using namespace std;

__attribute((unused)) static void gemm_CPU_reference_row_major(const float *A,
                                                               const float *B,
                                                               float *Result,
                                                               size_t M,
                                                               size_t N,
                                                               size_t K) {
// This is row-major
  for (size_t i = 0; i < M; i++) {
    for (size_t j = 0; j < N; j++) {
      float sum = 0;
      for (size_t k = 0; k < K; k++) {
        sum += A[i * K + k] * B[k * N + j];
      }
      Result[i * N + j] = sum;
    }
  }
}

static void gemm_CPU_reference(const float *A, const float *B, float *Result, size_t M, size_t N, size_t K) {
  // This is column-major
  for (size_t i = 0; i < M; i++) {
    for (size_t j = 0; j < N; j++) {
      float sum = 0;
      for (size_t k = 0; k < K; k++) {
        sum += A[k * M + i] * B[j * K + k];
      }
      Result[j * M + i] = sum;
    }
  }
}

void gemm_cuBLAS_reference(const float *A, const float *B, float *Result, size_t M, size_t N, size_t K) {
  float *d_A, *d_B, *d_Result;
  auto start = std::chrono::high_resolution_clock::now();
  hipMalloc(&d_A, M * K * sizeof(float));
  hipMalloc(&d_B, K * N * sizeof(float));
  hipMalloc(&d_Result, M * N * sizeof(float));

  hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasSgemm(handle,
              HIPBLAS_OP_N,
              HIPBLAS_OP_N,
              (int) M,
              (int) N,
              (int) K,
              &alpha,
              d_A,
              (int) N,
              d_B,
              (int) K,
              &beta,
              d_Result,
              (int) N);

  hipblasDestroy(handle);
  hipMemcpy(Result, d_Result, M * N * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_Result);

}

TEST(gemm, test_gemm_cuBLAS_param) {
  // Test if we entered the correct cuBLAS parameters
  auto *A = new float[256 * 128];
  auto *B = new float[128 * 256];

  // Randomly initialize A, B
  default_random_engine generator(RANDOM_SEED);
  uniform_real_distribution<float> matrix_dist(-1.0e2, 1.0e2);

  for (int i = 0; i < 128 * 256; i++) {
    A[i] = matrix_dist(generator);
    B[i] = matrix_dist(generator);
  }

  // Create float matrix C
  auto *C = new float[256 * 256];
  auto *C_cublas = new float[256 * 256];

  // Compute C = A * B
  gemm_CPU_reference(A, B, C, 256, 256, 128);
  gemm_cuBLAS_reference(A, B, C_cublas, 256, 256, 128);

  // Check if C is correct
  for (int i = 0; i < 256 * 256; i++) {
    EXPECT_NEAR(C[i], C_cublas[i], 1.0f);
  }

}

TEST(gemm, test_gemm_naive_basic) {
  // Create float matrices A, B
  auto *A = new float[256 * 256];
  auto *B = new float[256 * 256];
  auto *A_float_16 = new float_16[256 * 256];
  auto *B_float_16 = new float_16[256 * 256];

  // Randomly initialize A, B
  default_random_engine generator(RANDOM_SEED);
  uniform_real_distribution<float> matrix_dist(-1.0e2, 1.0e2);

  for (int i = 0; i < 256 * 256; i++) {
    // Make sure the matrix have the same float
    float_16 a_float_16 = __float2half(matrix_dist(generator));
    float_16 b_float_16 = __float2half(matrix_dist(generator));
    A[i] = __half2float(a_float_16);
    B[i] = __half2float(b_float_16);
    A_float_16[i] = a_float_16;
    B_float_16[i] = b_float_16;
  }

  // Create float matrix C
  auto *C = new float[256 * 256];
  auto *C_cublas = new float[256 * 256];

  // Compute C = A * B
  gemm_naive(A_float_16, B_float_16, C, 256, 256, 256);
  gemm_cuBLAS_reference(A, B, C_cublas, 256, 256, 256);

  // Check if C is correct
  for (int i = 0; i < 256 * 256; i++) {
    ASSERT_NEAR(C[i], C_cublas[i], 1.0f);
  }
}

TEST(gemm, test_gemm_naive_rectangular) {
  // Create float matrices A, B
  auto *A = new float[256 * 64];
  auto *B = new float[64 * 256];
  auto *A_float_16 = new float_16[256 * 64];
  auto *B_float_16 = new float_16[64 * 256];

  // Randomly initialize A, B
  default_random_engine generator(RANDOM_SEED);
  uniform_real_distribution<float> matrix_dist(-1.0e2, 1.0e2);

  for (int i = 0; i < 64 * 256; i++) {
    // Make sure the matrix have the same float
    float_16 a_float_16 = __float2half(matrix_dist(generator));
    float_16 b_float_16 = __float2half(matrix_dist(generator));

    A[i] = __half2float(a_float_16);
    B[i] = __half2float(b_float_16);

    A_float_16[i] = a_float_16;
    B_float_16[i] = b_float_16;
  }

  // Create float matrix C
  auto *C = new float[256 * 256];
  auto *C_cublas = new float[256 * 256];

  // Compute C = A * B
  gemm_naive(A_float_16, B_float_16, C, 256, 256, 64);
  gemm_cuBLAS_reference(A, B, C_cublas, 256, 256, 64);

  // Check if C is correct
  for (int i = 0; i < 256 * 256; i++) {
    ASSERT_NEAR(C[i], C_cublas[i], 1.0f);
  }
}

